/*
 * Author:      Hampton Ford
 * Github:      @shford
 * Status:      Incomplete
 *
 * Notes:       Max file size set at 4GiB b/c of Crypto Library limitations
 *
 * License:
 *
 *
 * CUDA_Driver_Version:                         11.7
 * CUDA Capability Major/Minor version number:  8.6
 *
 * Performance:
 * ~todo time
 *
 */

#include "tasks.cuh"

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

/*
 * set shared memory capacity to 0 - hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_name), hipFuncAttributePreferredSharedMemoryCarveout, 0);
 *
 * todo V5:
 *  Multi-Thread/Multi-Process - Initial & Final File I/O
 *  Multi-Thread/Multi-Process - Task 2
 *  Replace calloc inside read_file() w/ hipHostMalloc() to pin memory
 *  Replace size w/ strlen calls in host b/c no need to transfer that
 *  In kernel change from byte-by-byte copying to memcpyasync or at least by multiples of largest primitive
 */


int main()
{
    #if VALID_USER_INPUTS
        printf("Hash Collider - Starting Task 1...");
        task1();
        return 0;
    #else
        printf("Invalid user input macro evaluation... Please ensure CUDA Version, DeviceQuery Information, and User Variables are correct.\n");
        return -1;
    #endif // VALID_USER_INPUTS
}
